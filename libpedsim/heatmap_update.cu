#include "hip/hip_runtime.h"
// #include "ped_model.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cstdlib>
#include <iostream>
#include <cmath>
#include <omp.h>

__global__
void fadeHeat(int *d_heatmap, int size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size*size)
    {
        d_heatmap[index] = (int)round(d_heatmap[index] * 0.80);
    }
}

__global__
void heatIntensify(int *d_heatmap, int *x, int *y, int agent_size, int size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; 
    if (index >= 0 && index < agent_size)
    {
      atomicAdd(&d_heatmap[y[index]*size + x[index]], 40);
    }	
}

__global__
void setMaxHeat(int *d_heatmap, int size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size*size)
    {
      if (d_heatmap[index] >= 255)
      {
          d_heatmap[index] = 255;
      }
    }
}


__global__
void scaleHeatmap(int *d_heatmap, int *d_scaledHeatmap, int size, int cellSize)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < size*size)
    {
      for (int cellY = 0; cellY < cellSize; cellY++)
	    {
        int scaledBlock = (blockIdx.x * cellSize + cellY) * blockDim.x * cellSize;
        for (int cellX = 0; cellX < cellSize; cellX++)
        {
          int scaledThread = threadIdx.x * cellSize + cellX;
          // atomicExch(&d_scaledHeatmap[scaledBlock + scaledThread], d_heatmap[index]);
          d_scaledHeatmap[scaledBlock + scaledThread] = d_heatmap[index];
          // # if __CUDA_ARCH__>=200
          //   printf("cellY: %d, cellX: %d\n", cellY, cellX);
          // #endif
        }
	    }
    }
}


__global__
void blurHeatmap(int *d_scaledHeatmap, int *d_blurredHeatmap, int scaledSize)
{
   int WEIGHTSUM = 273;
    const int w[5][5] = {
    { 1, 4, 7, 4, 1 },
    { 4, 16, 26, 16, 4 },
    { 7, 26, 41, 26, 7 },
    { 4, 16, 26, 16, 4 },
    { 1, 4, 7, 4, 1 }
  };

     int i = blockIdx.x;// * blockDim.x;
     int j = threadIdx.x;
     int sum = 0;
     if(i >= 2 && i < scaledSize -2 && j >= 2 && j < scaledSize -2)
     for (int k = -2; k < 3; k++)
      {
        for (int l = -2; l < 3; l++)
         {
           sum += w[2 + k][2 + l] * d_scaledHeatmap[(i + k) * blockDimx.x + (j + l)];
         }
      }
     int value = sum / WEIGHTSUM;
     d_blurredHeatmap[i * blockDimx.x + j]/*[j]*/ = 0x00FF0000 | value << 24;
}


int THREADSPERBLOCK = 256;
// Updates the heatmap according to the agent positions
void updateHeatFade(int *heatmap, int SIZE)
{
  int THREADSPERBLOCK = 256;
  // std::cout << "Fade\n";
  int *d_heatmap;
  hipMalloc((void **)&d_heatmap, SIZE*SIZE*sizeof(int));
  // printf("after malloc: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpy(d_heatmap, heatmap, SIZE*SIZE*sizeof(int), hipMemcpyHostToDevice);
  // printf("after HostToDevice: %s\n", hipGetErrorString(hipGetLastError()));
  fadeHeat<<<((SIZE*SIZE)/THREADSPERBLOCK)+1,THREADSPERBLOCK>>>(d_heatmap, SIZE);
  // printf("after fadeHeat: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpy(heatmap, d_heatmap, SIZE*SIZE*sizeof(int), hipMemcpyDeviceToHost);
  // printf("after DeviceToHost: %s\n", hipGetErrorString(hipGetLastError()));
  hipFree(d_heatmap);
  // printf("after Free: %s\n", hipGetErrorString(hipGetLastError()));
}

void updateHeatIntensity(int *heatmap, int *x, int *y, int agent_size, int SIZE)
{
  // int THREADSPERBLOCK = 32;
  int *d_heatmap, *d_x, *d_y;
  // printf("before malloc: %s\n", hipGetErrorString(hipGetLastError()));
  hipMalloc((void **)&d_heatmap, SIZE*SIZE*sizeof(int));
  hipMalloc((void **)&d_x, agent_size*sizeof(int));
  hipMalloc((void **)&d_y, agent_size*sizeof(int));
  // printf("after malloc: %s\n", hipGetErrorString(hipGetLastError()));

  hipMemcpy(d_heatmap, heatmap, SIZE*SIZE*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_x, x, agent_size*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, agent_size*sizeof(int), hipMemcpyHostToDevice);
  // printf("after memcpy: %s\n", hipGetErrorString(hipGetLastError()));
  heatIntensify<<<((agent_size)/THREADSPERBLOCK)+1,THREADSPERBLOCK>>>(d_heatmap, d_x, d_y, agent_size, SIZE);
  // printf("after function: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpy(heatmap, d_heatmap, SIZE*SIZE*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(x, d_x, agent_size*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(y, d_y, agent_size*sizeof(int), hipMemcpyDeviceToHost);
  // printf("after memcpy: %s\n", hipGetErrorString(hipGetLastError()));
  hipFree(d_heatmap);
  hipFree(d_x);
  hipFree(d_y);
  // printf("after free: %s\n", hipGetErrorString(hipGetLastError()));
}

void updateSetMaxHeat(int *heatmap, int SIZE)
{
  int THREADSPERBLOCK = 256;
  int *d_heatmap;
  // printf("before malloc: %s\n", hipGetErrorString(hipGetLastError()));
  hipMalloc((void **)&d_heatmap, SIZE*SIZE*sizeof(int));
  // printf("after malloc: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpy(d_heatmap, heatmap, SIZE*SIZE*sizeof(int), hipMemcpyHostToDevice);
  // printf("after memcpy: %s\n", hipGetErrorString(hipGetLastError()));
  setMaxHeat<<<((SIZE*SIZE)/THREADSPERBLOCK)+1,THREADSPERBLOCK>>>(d_heatmap, SIZE);
  // printf("after function: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpy(heatmap, d_heatmap, SIZE*SIZE*sizeof(int), hipMemcpyDeviceToHost);
  // printf("after memcpy: %s\n", hipGetErrorString(hipGetLastError()));
  hipFree(d_heatmap);
  // printf("after free: %s\n", hipGetErrorString(hipGetLastError()));
}

void updateScaledHeatmap(int *heatmap, int *scaledHeatmap, int SIZE, int cellSize)
{
  int THREADSPERBLOCK = 1024;
  int scaledSize = SIZE*SIZE*cellSize*cellSize;
  int *d_heatmap;
  int *d_scaledHeatmap;
  // printf("before malloc1: %s\n", hipGetErrorString(hipGetLastError()));
  hipMalloc((void **)&d_heatmap, SIZE*SIZE*sizeof(int));
  // printf("after malloc1: %s\n", hipGetErrorString(hipGetLastError()));
  hipMalloc((void **)&d_scaledHeatmap, scaledSize*sizeof(int));
  // printf("after malloc2: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpy(d_heatmap, heatmap, SIZE*SIZE*sizeof(int), hipMemcpyHostToDevice);
  // printf("after memcpy1: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpy(d_scaledHeatmap, scaledHeatmap, scaledSize*sizeof(int), hipMemcpyHostToDevice);
  // printf("after memcpy2: %s\n", hipGetErrorString(hipGetLastError()));
  scaleHeatmap<<<(SIZE*SIZE)/THREADSPERBLOCK,THREADSPERBLOCK>>>(d_heatmap, d_scaledHeatmap, SIZE, cellSize);
  // printf("after scaleHeatmap: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpy(heatmap, d_heatmap, SIZE*SIZE*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(scaledHeatmap, d_scaledHeatmap, scaledSize*sizeof(int), hipMemcpyDeviceToHost);
  // printf("after memecpy: %s\n", hipGetErrorString(hipGetLastError()));
  hipFree(d_heatmap);  
  hipFree(d_scaledHeatmap);  
  // printf("after free: %s\n", hipGetErrorString(hipGetLastError()));
}

void updateBlurredHeatmap(int *scaledHeatmap, int *blurredHeatmap, int scaledSize)
{
  int THREADSPERBLOCK = 5120;
  int *d_scaledHeatmap;
  int *d_blurredHeatmap;
  printf("before malloc1: %s\n", hipGetErrorString(hipGetLastError()));
  hipMalloc((void **)&d_scaledHeatmap, scaledSize*scaledSize*sizeof(int));
  hipMalloc((void **)&d_scaledHeatmap, scaledSize*scaledSize*sizeof(int));
  printf("after malloc2: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpy(d_blurredHeatmap, blurredHeatmap, scaledSize*scaledSize*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_scaledHeatmap, scaledHeatmap, scaledSize*scaledSize*sizeof(int), hipMemcpyHostToDevice);
  printf("after memcpy2: %s\n", hipGetErrorString(hipGetLastError()));
  blurHeatmap<<<(scaledSize*scaledSize)/THREADSPERBLOCK,THREADSPERBLOCK>>>(d_blurredHeatmap, d_scaledHeatmap, scaledSize);
  printf("after scaleHeatmap: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpy(blurredHeatmap, d_blurredHeatmap, scaledSize*scaledSize*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(scaledHeatmap, d_scaledHeatmap, scaledSize*scaledSize*sizeof(int), hipMemcpyDeviceToHost);
  printf("after memecpy: %s\n", hipGetErrorString(hipGetLastError()));
  hipFree(d_blurredHeatmap);  
  hipFree(d_scaledHeatmap);  
  printf("after free: %s\n", hipGetErrorString(hipGetLastError()));  
}

// 	// Weights for blur filter
// 	const int w[5][5] = {
// 		{ 1, 4, 7, 4, 1 },
// 		{ 4, 16, 26, 16, 4 },
// 		{ 7, 26, 41, 26, 7 },
// 		{ 4, 16, 26, 16, 4 },
// 		{ 1, 4, 7, 4, 1 }
// 	};

// #define WEIGHTSUM 273
// 	// Apply gaussian blurfilter		       
// 	for (int i = 2; i < SCALED_SIZE - 2; i++)
// 	{
// 		for (int j = 2; j < SCALED_SIZE - 2; j++)
// 		{
// 			int sum = 0;
// 			for (int k = -2; k < 3; k++)
// 			{
// 				for (int l = -2; l < 3; l++)
// 				{
// 					sum += w[2 + k][2 + l] * scaled_heatmap[i + k][j + l];
// 				}
// 			}
// 			int value = sum / WEIGHTSUM;
// 			blurred_heatmap[i][j] = 0x00FF0000 | value << 24;
// 		}
// 	}
// }
