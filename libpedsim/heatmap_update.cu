#include "hip/hip_runtime.h"
#include "heatmap_update.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cstdlib>
#include <iostream>
#include <cmath>
#include <omp.h>




__global__
void fadeHeat(int *d_heatmap, int size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size*size)
    {
        d_heatmap[index] = (int)round(d_heatmap[index] * 0.80);
    }
}

__global__
void heatIntensify(int *d_heatmap, int *x, int *y, int agent_size, int size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; 
    if (index >= 0 && index < agent_size)
    {
      atomicAdd(&d_heatmap[y[index]*size + x[index]], 40);
    }	
}

__global__
void setMaxHeat(int *d_heatmap, int size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size*size)
    {
      if (d_heatmap[index] >= 255)
      {
          d_heatmap[index] = 255;
      }
    }
}


__global__
void scaleHeatmap(int *d_heatmap, int *d_scaledHeatmap, int size, int cellSize)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < size*size)
    {
      for (int cellY = 0; cellY < cellSize; cellY++)
	    {
        int scaledBlock = (blockIdx.x * cellSize + cellY) * blockDim.x * cellSize;
        for (int cellX = 0; cellX < cellSize; cellX++)
        {
          int scaledThread = threadIdx.x * cellSize + cellX;
          // atomicExch(&d_scaledHeatmap[scaledBlock + scaledThread], d_heatmap[index]);
          d_scaledHeatmap[scaledBlock + scaledThread] = d_heatmap[index];
          // # if __CUDA_ARCH__>=200
          //   printf("cellY: %d, cellX: %d\n", cellY, cellX);
          // #endif
        }
	    }
    }
}


__global__
void blurHeatmap(int *d_scaledHeatmap, int *d_blurredHeatmap, int scaledSize)
{
    int WEIGHTSUM = 273;
    const int w[5][5] = {
    { 1, 4, 7, 4, 1 },
    { 4, 16, 26, 16, 4 },
    { 7, 26, 41, 26, 7 },
    { 4, 16, 26, 16, 4 },
    { 1, 4, 7, 4, 1 }
  };

     int i = blockIdx.x * blockDim.x;
     int modI = (blockIdx.x * blockDim.x + threadIdx.x) % scaledSize;
     int j = threadIdx.x;
     int sum = 0;
     if(modI >= 2 && modI < scaledSize-2 && i+j >= 2*scaledSize && blockIdx.x < 5*scaledSize-2)
     for (int k = -2; k < 3; k++)
      {
        for (int l = -2; l < 3; l++)
         {       
           if(i+scaledSize*k >= 2*scaledSize && i+scaledSize*k < scaledSize*scaledSize-2*scaledSize)
           sum += w[2 + k][2 + l] * d_scaledHeatmap[(i + k) + (j + l)]; 
         }
      }
     int value = sum / WEIGHTSUM;    
     d_blurredHeatmap[i + j] = 0x00FF0000 | value << 24;
}


int THREADSPERBLOCK = 1024;
// Updates the heatmap according to the agent positions
void updateHeatFade(int *heatmap, int SIZE)
{
  int *d_heatmap;
  hipMalloc((void **)&d_heatmap, SIZE*SIZE*sizeof(int));
  // printf("after malloc: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpyAsync(d_heatmap, heatmap, SIZE*SIZE*sizeof(int), hipMemcpyHostToDevice);
  // printf("after HostToDevice: %s\n", hipGetErrorString(hipGetLastError()));
  fadeHeat<<<((SIZE*SIZE)/THREADSPERBLOCK)+1,THREADSPERBLOCK>>>(d_heatmap, SIZE);
  // printf("after fadeHeat: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpyAsync(heatmap, d_heatmap, SIZE*SIZE*sizeof(int), hipMemcpyDeviceToHost);
  // printf("after DeviceToHost: %s\n", hipGetErrorString(hipGetLastError()));
  hipFree(d_heatmap);
  // printf("after Free: %s\n", hipGetErrorString(hipGetLastError()));
}

void updateHeatIntensity(int *heatmap, int *x, int *y, int agent_size, int SIZE)
{
  int *d_heatmap, *d_x, *d_y;
  // printf("before malloc: %s\n", hipGetErrorString(hipGetLastError()));
  hipMalloc((void **)&d_heatmap, SIZE*SIZE*sizeof(int));
  hipMalloc((void **)&d_x, agent_size*sizeof(int));
  hipMalloc((void **)&d_y, agent_size*sizeof(int));
  // printf("after malloc: %s\n", hipGetErrorString(hipGetLastError()));

  hipMemcpyAsync(d_heatmap, heatmap, SIZE*SIZE*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_x, x, agent_size*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_y, y, agent_size*sizeof(int), hipMemcpyHostToDevice);
  // printf("after memcpy: %s\n", hipGetErrorString(hipGetLastError()));
  heatIntensify<<<((agent_size)/THREADSPERBLOCK)+1,THREADSPERBLOCK>>>(d_heatmap, d_x, d_y, agent_size, SIZE);
  // printf("after function: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpyAsync(heatmap, d_heatmap, SIZE*SIZE*sizeof(int), hipMemcpyDeviceToHost);
  // hipMemcpy(x, d_x, agent_size*sizeof(int), hipMemcpyDeviceToHost);
  // hipMemcpy(y, d_y, agent_size*sizeof(int), hipMemcpyDeviceToHost);
  // printf("after memcpy: %s\n", hipGetErrorString(hipGetLastError()));
  hipFree(d_heatmap);
  hipFree(d_x);
  hipFree(d_y);
  // printf("after free: %s\n", hipGetErrorString(hipGetLastError()));
}

void updateSetMaxHeat(int *heatmap, int SIZE)
{
  int *d_heatmap;
  // printf("before malloc: %s\n", hipGetErrorString(hipGetLastError()));
  hipMalloc((void **)&d_heatmap, SIZE*SIZE*sizeof(int));
  // printf("after malloc: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpyAsync(d_heatmap, heatmap, SIZE*SIZE*sizeof(int), hipMemcpyHostToDevice);
  // printf("after memcpy: %s\n", hipGetErrorString(hipGetLastError()));
  setMaxHeat<<<((SIZE*SIZE)/THREADSPERBLOCK)+1,THREADSPERBLOCK>>>(d_heatmap, SIZE);
  // printf("after function: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpyAsync(heatmap, d_heatmap, SIZE*SIZE*sizeof(int), hipMemcpyDeviceToHost);
  // printf("after memcpy: %s\n", hipGetErrorString(hipGetLastError()));
  hipFree(d_heatmap);
  // printf("after free: %s\n", hipGetErrorString(hipGetLastError()));
}

void updateScaledHeatmap(int *heatmap, int *scaledHeatmap, int SIZE, int cellSize)
{
  int scaledSizeSquared = SIZE*SIZE*cellSize*cellSize;
  int *d_heatmap;
  int *d_scaledHeatmap;
  // printf("before malloc1: %s\n", hipGetErrorString(hipGetLastError()));
  hipMalloc((void **)&d_heatmap, SIZE*SIZE*sizeof(int));
  hipMalloc((void **)&d_scaledHeatmap, scaledSizeSquared*sizeof(int));
  // printf("after malloc2: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpyAsync(d_heatmap, heatmap, SIZE*SIZE*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_scaledHeatmap, scaledHeatmap, scaledSizeSquared*sizeof(int), hipMemcpyHostToDevice);
  // printf("after memcpy2: %s\n", hipGetErrorString(hipGetLastError()));
  scaleHeatmap<<<(SIZE*SIZE)/THREADSPERBLOCK,THREADSPERBLOCK>>>(d_heatmap, d_scaledHeatmap, SIZE, cellSize);
  // printf("after scaleHeatmap: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpyAsync(heatmap, d_heatmap, SIZE*SIZE*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpyAsync(scaledHeatmap, d_scaledHeatmap, scaledSizeSquared*sizeof(int), hipMemcpyDeviceToHost);
  // printf("after memecpy: %s\n", hipGetErrorString(hipGetLastError()));
  hipFree(d_heatmap);  
  hipFree(d_scaledHeatmap);  
  // printf("after free: %s\n", hipGetErrorString(hipGetLastError()));
}

void updateBlurredHeatmap(int *scaledHeatmap, int *blurredHeatmap, int scaledSize)
{
  int *d_scaledHeatmap;
  int *d_blurredHeatmap;
  // printf("before malloc1: %s\n", hipGetErrorString(hipGetLastError()));
  hipMalloc((void **)&d_scaledHeatmap, scaledSize*scaledSize*sizeof(int));
  hipMalloc((void **)&d_blurredHeatmap, scaledSize*scaledSize*sizeof(int));
  // printf("after malloc2: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpyAsync(d_blurredHeatmap, blurredHeatmap, scaledSize*scaledSize*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpyAsync(d_scaledHeatmap, scaledHeatmap, scaledSize*scaledSize*sizeof(int), hipMemcpyHostToDevice);
  // printf("after memcpy2: %s\n", hipGetErrorString(hipGetLastError()));
  blurHeatmap<<<(scaledSize*scaledSize)/THREADSPERBLOCK,THREADSPERBLOCK>>>(d_scaledHeatmap, d_blurredHeatmap, scaledSize);
  // printf("after scaleHeatmap: %s\n", hipGetErrorString(hipGetLastError()));
  hipMemcpyAsync(blurredHeatmap, d_blurredHeatmap, scaledSize*scaledSize*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpyAsync(scaledHeatmap, d_scaledHeatmap, scaledSize*scaledSize*sizeof(int), hipMemcpyDeviceToHost);
  // printf("after memecpy: %s\n", hipGetErrorString(hipGetLastError()));
  hipFree(d_blurredHeatmap);  
  hipFree(d_scaledHeatmap);  
  // printf("after free: %s\n", hipGetErrorString(hipGetLastError()));  
}